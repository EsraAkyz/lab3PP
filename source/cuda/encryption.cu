#include "hip/hip_runtime.h"
#include "encryption.cuh"
#include "util/Hash.h"
#include "util/SharedMemory.h"

#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

/** Kernel of hash function **/
__global__ void hash(const std::uint64_t* const values, std::uint64_t* const hashes, const unsigned int length){
    // Calculate the global index of the current thread
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if thread is within length
    if(index < length){
        std::uint64_t hash_value = Hash::hash(values[index]);
        hashes[index] = hash_value;
    }
}

/** Kernel of flat_hash **/
#define FLAT_HASH_SHARED_MEM 64 // 128 or 64???
__global__ void flat_hash(const std::uint64_t* const values, std::uint64_t* const hashes, const unsigned int length){
    // Allocate shared memory
    __shared__ std::uint64_t shared_values[FLAT_HASH_SHARED_MEM_SIZE];
    __shared__ std::uint64_t shared_hashes[FLAT_HASH_SHARED_MEM_SIZE];

    // Calculate the global index of current thread
    unsigned int index = threadIdx.x; // Cause the kernel is invoked with (1, 1, 1) thread blocks of size (tx, 1, 1)

    // Check if thread within length
    if(index < length){
        // Load the value into shared memory
        shared_values[index] = values[index];
        __syncthreads();  // Ensure all threads have loaded their values

        // Calculate the hash value for the corresponding value
        shared_hashes[index] = Hash::hash(shared_values[index]);
        __syncthreads();  // Ensure all threads have calculated their hashes

        // Write the hash value to global memory
        hashes[index] = shared_hashes[index];
    }
}
