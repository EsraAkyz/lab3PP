#include "hip/hip_runtime.h"
#include "image.cuh"
#include "common.cuh"
#include "bitmap_image.h"

#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#incude <cstdint>

/** grayscale_kernel **/
__global__ void grayscale_kernel(const Pixel<std::uint8_t>* const input, Pixel<std::uint8_t>* const output,
                                 const unsigned int width, const unsigned int height){
    // Calculate the global index of the current thread
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if the thread is within the image dimensions
    if (idx < width && idy < height) {
        // Calculate the index of the current pixel
        unsigned int index = idy * width + idx;

        // Get the RGB values of the current pixel
        std::uint8_t r = input[index].get_red_channel();
        std::uint8_t g = input[index].get_green_channel();
        std::uint8_t b = input[index].get_blue_channel();

        // Calculate the grayscale value using the luminosity method
        // Y = 0.299 * R + 0.587 * G + 0.114 * B
        std::uint8_t grayscale = static_cast<std::uint8_t>(0.299f * r + 0.587f * g + 0.114f * b);

        // Set the grayscale value to the output pixel
        output[index] = { grayscale, grayscale, grayscale };
    }
}
